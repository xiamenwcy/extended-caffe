#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>


#include "caffe/layers/unpooling_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FixedUnPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height, const int width,
    const int unpooled_height, const int unpooled_width, const int out_kernel_h,
    const int out_kernel_w, const int out_stride_h, const int out_stride_w,
    const int out_pad_h, const int out_pad_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(unpool_index, nthreads) {
    int uw = unpool_index % unpooled_width;
    int uh = (unpool_index / unpooled_width) % unpooled_height;
    int c = (unpool_index / unpooled_width / unpooled_height) % channels;
    int n = unpool_index / unpooled_width / unpooled_height / channels;
    int hstart = (uh + out_pad_h < out_kernel_h) ? 0 :
      (uh + out_pad_h - out_kernel_h) / out_stride_h + 1;
    int hend = min((uh + out_pad_h) / out_stride_h + 1, height);
    int wstart = (uw + out_pad_w < out_kernel_w) ? 0 :
      (uw + out_pad_w - out_kernel_w) / out_stride_w + 1;
    int wend = min((uw + out_pad_w) / out_stride_w + 1, width);
    int offset = (n * channels + c) * height * width;
    int unpool_offset = (n * channels + c) * unpooled_height * unpooled_width;
    bottom_data += offset;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int uhstart = h * out_stride_h - out_pad_h;
        int uwstart = w * out_stride_w - out_pad_w;
        int uhend = uhstart + out_kernel_h;
        int uwend = uwstart + out_kernel_w;
        int uhmid = (uhstart + uhend - 1) / 2;
        int uwmid = (uwstart + uwend - 1) / 2;
        uhmid = min(max(uhmid, 0), unpooled_height);
        uwmid = min(max(uwmid, 0), unpooled_width);
        if (unpool_offset + uhmid * unpooled_width + uwmid == unpool_index) {
          // find the mapping, assign & return
          int index = h * width + w;
          top_data[unpool_index] = bottom_data[index];
          return;
        }
      }
    }
  }
}

template <typename Dtype>
__global__ void DivUnPoolForward(const int nthreads, const Dtype* bottom_data,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(unpool_index, nthreads) {
    int uw = unpool_index % unpooled_width + out_pad_w;
    int uh = (unpool_index / unpooled_width) % unpooled_height + out_pad_h;
    int c = (unpool_index / unpooled_width / unpooled_height) % channels;
    int n = unpool_index / unpooled_width / unpooled_height / channels;
    int spatial_dim = unpooled_height * unpooled_width;
    int hstart = (uh < out_kernel_h) ? 0 :
      (uh - out_kernel_h) / out_stride_h + 1;
    int hend = min(uh / out_stride_h + 1, height);
    int wstart = (uw < out_kernel_w) ? 0 : 
      (uw - out_kernel_w) / out_stride_w + 1;
    int wend = min(uw / out_stride_w + 1, width);
    Dtype divval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int uhstart = h * out_stride_h - out_pad_h;
        int uwstart = w * out_stride_w - out_pad_w;
        int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
        int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
        int unpool_size = (uhend - uhstart) * (uwend - uwstart);
        divval += bottom_data[h * width + w] / unpool_size;
      }
    }
    top_data[unpool_index] = divval / mask[unpool_index % spatial_dim];
  }
}

template <typename Dtype>
__global__ void RepUnPoolForward(const int nthreads, const Dtype* bottom_data,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(unpool_index, nthreads) {
    int uw = unpool_index % unpooled_width + out_pad_w;
    int uh = (unpool_index / unpooled_width) % unpooled_height + out_pad_h;
    int c = (unpool_index / unpooled_width / unpooled_height) % channels;
    int n = unpool_index / unpooled_width / unpooled_height / channels;
    int spatial_dim = unpooled_height * unpooled_width;
    int hstart = (uh < out_kernel_h) ? 0 :
      (uh - out_kernel_h) / out_stride_h + 1;
    int hend = min(uh / out_stride_h + 1, height);
    int wstart = (uw < out_kernel_w) ? 0 : 
      (uw - out_kernel_w) / out_stride_w + 1;
    int wend = min(uw / out_stride_w + 1, width);
    Dtype val = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int uhstart = h * out_stride_h - out_pad_h;
        int uwstart = w * out_stride_w - out_pad_w;
        int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
        int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
        val += bottom_data[h * width + w];
      }
    }
    top_data[unpool_index] = val / mask[unpool_index % spatial_dim];
  }
}

template <typename Dtype>
void UnPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = top[0]->count();
  caffe_gpu_set(count, Dtype(0), top_data);
  const int* mask = mask_.gpu_data();
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnPoolingParameter_UnPoolMethod_FIXED:
    // NOLINT_NEXT_LINE(whitespace/operators)
    FixedUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, top_data);
    break;
  case UnPoolingParameter_UnPoolMethod_DIV:
    // NOLINT_NEXT_LINE(whitespace/operators)
    DivUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, top_data);
    break;
  case UnPoolingParameter_UnPoolMethod_REP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    RepUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, top_data);
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void FixedUnPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int uhstart = h * out_stride_h - out_pad_h;
    int uwstart = w * out_stride_w - out_pad_w;
    int uhend = uhstart + out_kernel_h;
    int uwend = uwstart + out_kernel_w;
    int uhmid = (uhstart + uhend - 1) / 2;
    int uwmid = (uwstart + uwend - 1) / 2;
    uhmid = min(max(uhmid, 0), unpooled_height-1);
    uwmid = min(max(uwmid, 0), unpooled_width-1);
    int offset = (n * channels + c) * unpooled_height * unpooled_width;
    int unpool_index = uhmid * unpooled_width + uwmid;
    Dtype gradient = 0;
    if (mask[unpool_index] == h * width + w) {
      gradient += top_diff[unpool_index + offset];
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void DivUnPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int uhstart = h * out_stride_h - out_pad_h;
    int uwstart = w * out_stride_w - out_pad_w;
    int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
    int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
    int unpool_size = (uhend - uhstart) * (uwend - uwstart);
    uhstart = max(uhstart, 0);
    uwstart = max(uwstart, 0);
    uhend = min(uhend, unpooled_height);
    uwend = min(uwend, unpooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * unpooled_height * unpooled_width;
    for (int uh = uhstart; uh < uhend; ++uh) {
      for (int uw = uwstart; uw < uwend; ++uw) {
        int unpool_index = uh * unpooled_width + uw;
        gradient += top_diff[unpool_index + offset] / mask[unpool_index];
      }
    }
    bottom_diff[index] = gradient / unpool_size;
  }
}

template <typename Dtype>
__global__ void RepUnPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int out_kernel_h, const int out_kernel_w, const int out_stride_h,
    const int out_stride_w, const int out_pad_h, const int out_pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int uhstart = h * out_stride_h - out_pad_h;
    int uwstart = w * out_stride_w - out_pad_w;
    int uhend = min(uhstart + out_kernel_h, unpooled_height + out_pad_h);
    int uwend = min(uwstart + out_kernel_w, unpooled_width + out_pad_w);
    uhstart = max(uhstart, 0);
    uwstart = max(uwstart, 0);
    uhend = min(uhend, unpooled_height);
    uwend = min(uwend, unpooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * unpooled_height * unpooled_width;
    for (int uh = uhstart; uh < uhend; ++uh) {
      for (int uw = uwstart; uw < uwend; ++uw) {
        int unpool_index = uh * unpooled_width + uw;
        gradient += top_diff[unpool_index + offset] / mask[unpool_index];
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void UnPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  const int* mask = mask_.gpu_data();
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnPoolingParameter_UnPoolMethod_FIXED:
    // NOLINT_NEXT_LINE(whitespace/operators)
    FixedUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, bottom_diff);
    break;
  case UnPoolingParameter_UnPoolMethod_DIV:
    // NOLINT_NEXT_LINE(whitespace/operators)
    DivUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, bottom_diff);
    break;
  case UnPoolingParameter_UnPoolMethod_REP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    RepUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, num_, channels_, height_, width_,
        unpooled_height_, unpooled_width_, out_kernel_h_, out_kernel_w_,
        out_stride_h_, out_stride_w_, out_pad_h_, out_pad_w_, bottom_diff);
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UnPoolingLayer);


}  // namespace caffe
